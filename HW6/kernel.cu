#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

#define KERNEL_NUM 0

#define RUN_COUNT 10
#define BLOCK_SIZE_EXP 10

float computeTimeSum = 0, totalTimeSum = 0;

void constantInit(int *data, int size, int val) {
	for (int i = 0; i < size; ++i) {
		data[i] = val;
	}
}

void executeKernel(dim3 gridDim, dim3 blockDim, int *inputData, int *outputData) {
	#if KERNEL_NUM == 1
		matMulA1Kernel <<<gridDim, blockDim>>> (d_C, d_A, d_B, n);
	#elif KERNEL_NUM == 2
		matMulA2Kernel <<<gridDim, blockDim>>> (d_C, d_A, d_B, n);
	#elif KERNEL_NUM == 3
		matMulA3Kernel <<<gridDim, blockDim>>> (d_C, d_A, d_B, n);
	#elif KERNEL_NUM == 4
		matMulA4Kernel <<<gridDim, blockDim>>> (d_C, d_A, d_B, n);
	#elif KERNEL_NUM == 5
		matMulA4Kernel <<<gridDim, blockDim>>> (d_C, d_A, d_B, n);
	#endif
}

void reduce(int exp) {
    int n = 1 << exp;
	size_t mem_size = sizeof(int) * n;

    int rounds = (exp + BLOCK_SIZE_EXP - 1) / BLOCK_SIZE_EXP;

	hipError_t error;

	// Allocate host memory for matrices A and B
	int *h_A;
    error = hipHostMalloc(&h_A, mem_size);
    if (error != hipSuccess) {
        printf("[!] A allocation failed!\n");
        exit(1);
    }
	constantInit(h_A, n, 1);

	// Allocate CUDA events that we'll use for timing
	hipEvent_t copyStart, copyStop, computeStart, computeStop;
	error = hipEventCreate(&copyStart);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to create copyStart event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	error = hipEventCreate(&copyStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to create copyStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	error = hipEventCreate(&computeStart);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to create computeStart event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	error = hipEventCreate(&computeStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to create computeStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(copyStart, NULL);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to record copyStart event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Allocate device memory
	int *dev_A;
	error = hipMalloc((void **)&dev_A, mem_size);
	if (error != hipSuccess) {
		printf("hipMalloc dev_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(dev_A, h_A, mem_size, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("hipMemcpy (dev_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(computeStart, NULL);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to record computeStart event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	const int BLOCK_SIZE = 1 << BLOCK_SIZE_EXP;
	dim3 gridDim, blockDim;
	int *inputData = dev_A, *outputData;
	for (int round = rounds; round > 0; round--, exp -= BLOCK_SIZE_EXP, n = 1 << exp) {
		if (round == 1) { // it's the last round
			blockDim = dim3(n, 1, 1);
			gridDim = dim3(1, 1, 1);
		} else {
			blockDim = dim3(BLOCK_SIZE, 1, 1);
			gridDim = dim3(1 << (exp - BLOCK_SIZE_EXP), 1, 1);
		}
		printf("blocks: (%d, %d, %d), grid(%d, %d, %d), exp: %d, n: %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, exp, n);

		// Execute the kernel
		outputData = hipMalloc(&outputData, size);
		executeKernel(gridDim, blockDim, inputData, outputData);
		hipFree(inputData);
		inputData = outputData;
	}
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	error = hipEventRecord(computeStop, NULL);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to record computeStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(computeStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to synchronize on the computeStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	// error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
	// if (error != hipSuccess) {
	// 	printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	// 	exit(EXIT_FAILURE);
	// }

	error = hipEventRecord(copyStop, NULL);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to record copyStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	error = hipEventSynchronize(copyStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to synchronize on the copyStop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float totalMsec, computeMsec;
	error = hipEventElapsedTime(&totalMsec, copyStart, copyStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to get time elapsed between copy events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	error = hipEventElapsedTime(&computeMsec, computeStart, computeStop);
	if (error != hipSuccess) {
		fprintf(stderr, "Failed to get time elapsed between compute events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	printf("Total time = %f, compute time = %f (ms)\n", totalMsec, computeMsec);
	totalTimeSum += totalMsec;
	computeTimeSum += computeMsec;

	// Clean up memory
	hipHostFree(h_A);
	hipFree(dev_A);
}


/**
* Program main
*/
int main(int argc, char **argv)
{
	// Size of square matrices
	unsigned short exp = 0;
	printf("[-] Exponent = ");
	scanf("%u", &exp);
    long n = 1 << exp;

    printf("Reducing %ld size array.\n", n);

	for (int i = 0; i < RUN_COUNT; i++) {
        reduce(exp);
	}

    float avg = computeTimeSum / RUN_COUNT;
	printf("\n[-] Average total time = %f , average compute time = %f\n", totalTimeSum / RUN_COUNT, avg);
    printf("[-] Average memory bandwidth = %f GB/s\n", n * 4.0 / (avg / 1000) / (1 << 30));

	return 0;
}
